#include "hip/hip_runtime.h"
/* +++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
   Copyright (c) 2023 Daniele Rapetti

   This file is part of cudaOnPlumed.

   cudaOnPlumed is free software: you can redistribute it and/or modify
   it under the terms of the GNU Lesser General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.

   cudaOnPlumed is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU Lesser General Public License for more details.

   You should have received a copy of the GNU Lesser General Public License
   along with cudaOnPlumed.  If not, see <http://www.gnu.org/licenses/>.
+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++ */
#include "plumed/colvar/CoordinationBase.h"
#include "plumed/core/ActionRegister.h"
#include "plumed/tools/NeighborList.h"
#include "plumed/tools/SwitchingFunction.h"

#include "cudaHelpers.cuh"
// #include "ndReduction.h"

#include <cub/block/block_load.cuh>
#include <cub/block/block_reduce.cuh>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "hip/hip_runtime.h"
#include ""

// cfloat for DLB_EPSILON and FLT_EPSILON
#include <cfloat>

#include <iostream>
#include <limits>
#include <numeric>

using std::cerr;

// #define vdbg(...) std::cerr << __LINE__ << ":" << #__VA_ARGS__ << " " <<
// (__VA_ARGS__) << '\n'
#define vdbg(...)

namespace PLMD {
namespace colvar {
//+PLUMEDOC COLVAR CUDACOORDINATION
/*
Calculate coordination numbers. Like coordination, but on nvdia gpu and with
limited switching.

CUDACOORDINATION can be invoked with CUDACOORDINATIONFLOAT, but that version
will use single floating point precision, while being faster and compatible with
desktop-based Nvidia cards.

This keyword can be used to calculate the number of contacts between two groups
of atoms and is defined as \f[ \sum_{i\in A} \sum_{i\in B} s_{ij} \f] where
\f$s_{ij}\f$ is 1 if the contact between atoms \f$i\f$ and \f$j\f$ is formed,
zero otherwise.
In actuality, \f$s_{ij}\f$ is replaced with a switching function so as to ensure
that the calculated CV has continuous derivatives. The default switching
function is: \f[ s_{ij} = \frac{ 1 - \left(\frac{{\bf r}_{ij}}{r_0}\right)^n
} { 1 - \left(\frac{{\bf r}_{ij}}{r_0}\right)^m } \f].


\par Examples

Here's an example that shows what happens when providing COORDINATION with
a single group:
\plumedfile
# define some huge group:
group: GROUP ATOMS=1-1000
# Here's coordination within a single group:
CUDACOORDINATION GROUPA=group R_0=0.3

\endplumedfile

*/
//+ENDPLUMEDOC

// these constant will be used within the kernels
template <typename calculateFloat> struct rationalSwitchParameters {
  calculateFloat dmaxSQ = std::numeric_limits<calculateFloat>::max();
  calculateFloat invr0_2 = 1.0; // r0=1
  calculateFloat stretch = 1.0;
  calculateFloat shift = 0.0;
  int nn = 6;
  int mm = 12;
};

template <typename calculateFloat> struct ortoPBCs {
  calculateFloat invX = 1.0;
  calculateFloat invY = 1.0;
  calculateFloat invZ = 1.0;
  calculateFloat X = 1.0;
  calculateFloat Y = 1.0;
  calculateFloat Z = 1.0;
};

template <typename calculateFloat>
__device__ calculateFloat pbcClamp(calculateFloat x) {
  return 0.0;
}

template <> __device__ __forceinline__ double pbcClamp<double>(double x) {
  // convert a double to a signed int in round-to-nearest-even mode.
  return __double2int_rn(x) - x;
  // return x - floor(x+0.5);
  // Round argument x to an integer value in single precision floating-point
  // format.
  // Uses round to nearest rounding, with ties rounding to even.
  // return nearbyint(x) - x;
}

template <> __device__ __forceinline__ float pbcClamp<float>(float x) {
  // convert a double to a signed int in round-to-nearest-even mode.
  return __float2int_rn(x) - x;
  // return x - floorf(x+0.5f);
  // return nearbyintf(x) - x;
}

// does not inherit from coordination base because nl is private
template <typename calculateFloat> class CudaCoordination : public Colvar {
  /// the pointer to the coordinates on the GPU
  thrust::device_vector<calculateFloat> cudaPositions;
  /// the pointer to the nn list on the GPU
  thrust::device_vector<calculateFloat> cudaCoordination;
  thrust::device_vector<calculateFloat> cudaDerivatives;
  thrust::device_vector<calculateFloat> cudaVirial;
  thrust::device_vector<calculateFloat> reductionMemoryVirial;
  thrust::device_vector<calculateFloat> reductionMemoryCoord;
  thrust::device_vector<unsigned> cudaTrueIndexes;

  hipStream_t streamDerivatives;
  hipStream_t streamVirial;
  hipStream_t streamCoordination;

  unsigned maxNumThreads = 512;
  rationalSwitchParameters<calculateFloat> switchingParameters;
  ortoPBCs<calculateFloat> myPBC;

  bool pbc{true};
  void setUpPermanentGPUMemory();

public:
  explicit CudaCoordination(const ActionOptions &);
  virtual ~CudaCoordination();
  // active methods:
  static void registerKeywords(Keywords &keys);
  void calculate() override;
};
using CudaCoordination_d = CudaCoordination<double>;
using CudaCoordination_f = CudaCoordination<float>;
PLUMED_REGISTER_ACTION(CudaCoordination_d, "CUDACOORDINATION")
PLUMED_REGISTER_ACTION(CudaCoordination_f, "CUDACOORDINATIONFLOAT")

template <typename calculateFloat>
void CudaCoordination<calculateFloat>::setUpPermanentGPUMemory() {
  auto nat = getPositions().size();
  cudaPositions.resize(3 * nat);
  cudaDerivatives.resize(3 * nat);
  cudaTrueIndexes.resize(nat);
  std::vector<unsigned> trueIndexes(nat);
  for (size_t i = 0; i < nat; ++i) {
    trueIndexes[i] = getAbsoluteIndex(i).index();
  }
  cudaTrueIndexes = trueIndexes;
}

template <typename calculateFloat>
void CudaCoordination<calculateFloat>::registerKeywords(Keywords &keys) {
  Colvar::registerKeywords(keys);

  keys.add("optional", "THREADS", "The upper limit of the number of threads");
  keys.add("atoms", "GROUPA", "First list of atoms");

  keys.add("compulsory", "NN", "6",
           "The n parameter of the switching function ");
  keys.add("compulsory", "MM", "0",
           "The m parameter of the switching function; 0 implies 2*NN");
  keys.add("compulsory", "R_0", "The r_0 parameter of the switching function");
  keys.add("compulsory", "D_MAX", "0.0",
           "The cut off of the switching function");
}

template <typename calculateFloat>
__device__ __forceinline__ calculateFloat pcuda_fastpow(calculateFloat base,
                                                        int expo) {
  if (expo < 0) {
    expo = -expo;
    base = 1.0 / base;
  }
  calculateFloat result = 1.0;
  while (expo) {
    if (expo & 1)
      result *= base;
    expo >>= 1;
    base *= base;
  }
  return result;
}

template <typename calculateFloat> __device__ calculateFloat pcuda_eps() {
  return 0;
}

template <> constexpr __device__ float pcuda_eps<float>() {
  return FLT_EPSILON * 10.0f;
}
template <> constexpr __device__ double pcuda_eps<double>() {
  return DBL_EPSILON * 10.0;
}

template <typename calculateFloat>
__device__ __forceinline__ calculateFloat
pcuda_Rational(const calculateFloat rdist, const int NN, const int MM,
               calculateFloat &dfunc) {
  calculateFloat result;
  if (2 * NN == MM) {
    // if 2*N==M, then (1.0-rdist^N)/(1.0-rdist^M) = 1.0/(1.0+rdist^N)
    calculateFloat rNdist = pcuda_fastpow(rdist, NN - 1);
    result = 1.0 / (1 + rNdist * rdist);
    dfunc = -NN * rNdist * result * result;
  } else {
    if (rdist > (1. - pcuda_eps<calculateFloat>()) &&
        rdist < (1 + pcuda_eps<calculateFloat>())) {

      result = NN / MM;
      dfunc = 0.5 * NN * (NN - MM) / MM;
    } else {
      calculateFloat rNdist = pcuda_fastpow(rdist, NN - 1);
      calculateFloat rMdist = pcuda_fastpow(rdist, MM - 1);
      calculateFloat num = 1. - rNdist * rdist;
      calculateFloat iden = 1.0 / (1.0 - rMdist * rdist);
      result = num * iden;
      dfunc = ((-NN * rNdist * iden) + (result * (iden * MM) * rMdist));
    }
  }
  return result;
}

template <typename calculateFloat>
__global__ void getpcuda_Rational(const calculateFloat *rdists, const int NN,
                                  const int MM, calculateFloat *dfunc,
                                  calculateFloat *res) {
  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (rdists[i] <= 0.) {
    res[i] = 1.;
    dfunc[i] = 0.0;
  } else
    res[i] = pcuda_Rational(rdists[i], NN, MM, dfunc[i]);
  // printf("stretch: %i: %f -> %f\n",i,rdists[i],res[i]);
}

// __global__ void getConst() {
//   printf("Cuda: cu_epsilon = %f\n", cu_epsilon);
// }

template <typename calculateFloat>
CudaCoordination<calculateFloat>::CudaCoordination(const ActionOptions &ao)
    : PLUMED_COLVAR_INIT(ao) {
  std::vector<AtomNumber> GroupA;
  parseAtomList("GROUPA", GroupA);

  bool nopbc = !pbc;
  parseFlag("NOPBC", nopbc);
  pbc = !nopbc;

  parse("THREADS", maxNumThreads);
  if (maxNumThreads <= 0)
    error("THREADS should be positive");
  addValueWithDerivatives();
  setNotPeriodic();
  requestAtoms(GroupA);

  log.printf("  \n");
  if (pbc)
    log.printf("  using periodic boundary conditions\n");
  else
    log.printf("  without periodic boundary conditions\n");

  std::string sw, errors;

  { // loading data to the GPU
    int nn_ = 6;
    int mm_ = 0;

    calculateFloat r0_ = 0.0;
    parse("R_0", r0_);
    if (r0_ <= 0.0) {
      error("R_0 should be explicitly specified and positive");
    }

    parse("NN", nn_);
    parse("MM", mm_);
    if (mm_ == 0) {
      mm_ = 2 * nn_;
    }
    if (mm_ % 2 != 0 || mm_ % 2 != 0)
      error(" this implementation only works with both MM and NN even");

    switchingParameters.nn = nn_;
    switchingParameters.mm = mm_;
    switchingParameters.stretch = 1.0;
    switchingParameters.shift = 0.0;

    calculateFloat dmax = 0.0;
    parse("D_MAX", dmax);
    if (dmax == 0.0) { // TODO:check for a "non present flag"
      // set dmax to where the switch is ~0.00001
      dmax = r0_ * std::pow(0.00001, 1.0 / (nn_ - mm_));
      // ^This line is equivalent to:
      // SwitchingFunction tsw;
      // tsw.set(nn_,mm_,r0_,0.0);
      // dmax=tsw.get_dmax();
    }

    switchingParameters.dmaxSQ = dmax * dmax;
    calculateFloat invr0 = 1.0 / r0_;
    switchingParameters.invr0_2 = invr0 * invr0;
    constexpr bool dostretch = true;
    if (dostretch) {
      std::vector<calculateFloat> inputs = {0.0, dmax * invr0};

      thrust::device_vector<calculateFloat> inputZeroMax(2);
      inputZeroMax = inputs;
      thrust::device_vector<calculateFloat> dummydfunc(2);
      thrust::device_vector<calculateFloat> resZeroMax(2);

      getpcuda_Rational<<<1, 2>>>(thrust::raw_pointer_cast(inputZeroMax.data()),
                                  nn_, mm_,
                                  thrust::raw_pointer_cast(dummydfunc.data()),
                                  thrust::raw_pointer_cast(resZeroMax.data()));

      switchingParameters.stretch = 1.0 / (resZeroMax[0] - resZeroMax[1]);
      switchingParameters.shift = -resZeroMax[1] * switchingParameters.stretch;
    }
  }

  checkRead();
  hipStreamCreate(&streamDerivatives);
  hipStreamCreate(&streamVirial);
  hipStreamCreate(&streamCoordination);
  setUpPermanentGPUMemory();

  log << "  contacts are counted with cutoff (dmax)="
      << sqrt(switchingParameters.dmaxSQ)
      << ", with a rational switch with parameters: d0=0.0, r0="
      << 1.0 / sqrt(switchingParameters.invr0_2)
      << ", N=" << switchingParameters.nn << ", M=" << switchingParameters.mm
      << ".\n";
}

template <typename calculateFloat>
CudaCoordination<calculateFloat>::~CudaCoordination() {
  hipStreamDestroy(streamDerivatives);
  hipStreamDestroy(streamVirial);
  hipStreamDestroy(streamCoordination);
}

template <typename calculateFloat>
__device__ __forceinline__ calculateFloat
calculateSqr(const calculateFloat distancesq,
             const rationalSwitchParameters<calculateFloat> switchingParameters,
             calculateFloat &dfunc) {
  calculateFloat result = 0.0;
  dfunc = 0.0;
  if (distancesq < switchingParameters.dmaxSQ) {
    const calculateFloat rdist_2 = distancesq * switchingParameters.invr0_2;
    result = pcuda_Rational(rdist_2, switchingParameters.nn / 2,
                            switchingParameters.mm / 2, dfunc);
    // chain rule:
    dfunc *= 2 * switchingParameters.invr0_2;
    // cu_stretch:
    result = result * switchingParameters.stretch + switchingParameters.shift;
    dfunc *= switchingParameters.stretch;
  }
  return result;
}

#define X(I) 3 * I
#define Y(I) 3 * I + 1
#define Z(I) 3 * I + 2

template <bool usePBC, typename calculateFloat>
__global__ void
getCoord(const unsigned nat,
         const rationalSwitchParameters<calculateFloat> switchingParameters,
         const ortoPBCs<calculateFloat> myPBC,
         const calculateFloat *coordinates, const unsigned *trueIndexes,
         calculateFloat *ncoordOut, calculateFloat *devOut,
         calculateFloat *virialOut) {
  // blockDIm are the number of threads in your block
  const unsigned i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i >= nat) { // blocks are initializated with 'ceil (nat/threads)'
    return;
  }
  // we try working with less global memory possible, so we set up a bunch of
  // temporary variables
  const unsigned idx = trueIndexes[i];
  // local results
  calculateFloat mydevX = 0.0;
  calculateFloat mydevY = 0.0;
  calculateFloat mydevZ = 0.0;
  calculateFloat mycoord = 0.0;
  // the previous version used static array for myVirial and d
  // using explicit variables guarantees that this data will be stored in
  // registers
  calculateFloat myVirial_0 = 0.0;
  calculateFloat myVirial_1 = 0.0;
  calculateFloat myVirial_2 = 0.0;
  calculateFloat myVirial_3 = 0.0;
  calculateFloat myVirial_4 = 0.0;
  calculateFloat myVirial_5 = 0.0;
  calculateFloat myVirial_6 = 0.0;
  calculateFloat myVirial_7 = 0.0;
  calculateFloat myVirial_8 = 0.0;
  // local calculation aid
  calculateFloat x = coordinates[X(i)];
  calculateFloat y = coordinates[Y(i)];
  calculateFloat z = coordinates[Z(i)];
  calculateFloat d_0, d_1, d_2;
  calculateFloat t_0, t_1, t_2;
  calculateFloat dfunc;
  calculateFloat coord;
  for (unsigned j = 0; j < nat; ++j) {
    // const unsigned j = threadIdx.y + blockIdx.y * blockDim.y;

    // Safeguard
    if (idx == trueIndexes[j])
      continue;
    // or may be better to set up an
    // const unsigned xyz = threadIdx.z
    // where the third dim is 0 1 2 ^
    // ?
    if constexpr (usePBC) {
      d_0 = pbcClamp((coordinates[X(j)] - x) * myPBC.invX) * myPBC.X;
      d_1 = pbcClamp((coordinates[Y(j)] - y) * myPBC.invY) * myPBC.Y;
      d_2 = pbcClamp((coordinates[Z(j)] - z) * myPBC.invZ) * myPBC.Z;
    } else {
      d_0 = coordinates[X(j)] - x;
      d_1 = coordinates[Y(j)] - y;
      d_2 = coordinates[Z(j)] - z;
    }

    dfunc = 0.;
    coord = calculateSqr(d_0 * d_0 + d_1 * d_1 + d_2 * d_2, switchingParameters,
                         dfunc);

    t_0 = dfunc * d_0;
    t_1 = dfunc * d_1;
    t_2 = dfunc * d_2;
    mydevX -= t_0;
    mydevY -= t_1;
    mydevZ -= t_2;
    if (i < j) {
      mycoord += coord;
      myVirial_0 -= t_0 * d_0;
      myVirial_1 -= t_0 * d_1;
      myVirial_2 -= t_0 * d_2;
      myVirial_3 -= t_1 * d_0;
      myVirial_4 -= t_1 * d_1;
      myVirial_5 -= t_1 * d_2;
      myVirial_6 -= t_2 * d_0;
      myVirial_7 -= t_2 * d_1;
      myVirial_8 -= t_2 * d_2;
    }
  }
  // working in global memory ONLY at the end
  devOut[X(i)] = mydevX;
  devOut[Y(i)] = mydevY;
  devOut[Z(i)] = mydevZ;
  ncoordOut[i] = mycoord;
  virialOut[nat * 0 + i] = myVirial_0;
  virialOut[nat * 1 + i] = myVirial_1;
  virialOut[nat * 2 + i] = myVirial_2;
  virialOut[nat * 3 + i] = myVirial_3;
  virialOut[nat * 4 + i] = myVirial_4;
  virialOut[nat * 5 + i] = myVirial_5;
  virialOut[nat * 6 + i] = myVirial_6;
  virialOut[nat * 7 + i] = myVirial_7;
  virialOut[nat * 8 + i] = myVirial_8;
}

#define getCoordOrthoPBC getCoord<true>
#define getCoordNoPBC getCoord<false>

template <typename calculateFloat>
void CudaCoordination<calculateFloat>::calculate() {
  constexpr unsigned dataperthread = 4;
  auto positions = getPositions();
  auto nat = positions.size();
  /***************************copying data on the GPU**************************/
  CUDAHELPERS::plmdDataToGPU(cudaPositions, positions, streamDerivatives);
  /***************************copying data on the GPU**************************/

  Tensor virial;
  double coordination;
  auto deriv = std::vector<Vector>(nat);

  // constexpr unsigned nthreads = 512;

  unsigned ngroups = ceil(double(nat) / maxNumThreads);

  /**********************allocating the memory on the GPU**********************/
  cudaCoordination.resize(nat);
  cudaVirial.resize(nat * 9);
  /**************************starting the calculations*************************/
  // this calculates the derivatives and prepare the coordination and the
  // virial for the accumulation
  if (pbc) {
    // Only ortho as now
    auto box = getBox();

    myPBC.X = box(0, 0);
    myPBC.Y = box(1, 1);
    myPBC.Z = box(2, 2);
    myPBC.invX = 1.0 / myPBC.X;
    myPBC.invY = 1.0 / myPBC.Y;
    myPBC.invZ = 1.0 / myPBC.Z;

    getCoordOrthoPBC<<<ngroups, maxNumThreads, 0, streamDerivatives>>>(
        nat, switchingParameters, myPBC,
        thrust::raw_pointer_cast(cudaPositions.data()),
        thrust::raw_pointer_cast(cudaTrueIndexes.data()),
        thrust::raw_pointer_cast(cudaCoordination.data()),
        thrust::raw_pointer_cast(cudaDerivatives.data()),
        thrust::raw_pointer_cast(cudaVirial.data()));
  } else {
    getCoordNoPBC<<<ngroups, maxNumThreads, 0, streamDerivatives>>>(
        nat, switchingParameters, myPBC,
        thrust::raw_pointer_cast(cudaPositions.data()),
        thrust::raw_pointer_cast(cudaTrueIndexes.data()),
        thrust::raw_pointer_cast(cudaCoordination.data()),
        thrust::raw_pointer_cast(cudaDerivatives.data()),
        thrust::raw_pointer_cast(cudaVirial.data()));
  }

  /**************************accumulating the results**************************/

  hipDeviceSynchronize();

  CUDAHELPERS::plmdDataFromGPU(cudaDerivatives, deriv, streamDerivatives);

  auto N = nat;

  while (N > 1) {
    size_t runningThreads = CUDAHELPERS::threadsPerBlock(
        ceil(double(N) / dataperthread), maxNumThreads);

    unsigned nGroups = ceil(double(N) / (runningThreads * dataperthread));

    reductionMemoryVirial.resize(9 * nGroups);
    reductionMemoryCoord.resize(nGroups);

    dim3 ngroupsVirial(nGroups, 9);
    CUDAHELPERS::doReductionND<dataperthread>(
        thrust::raw_pointer_cast(cudaVirial.data()),
        thrust::raw_pointer_cast(reductionMemoryVirial.data()), N,
        ngroupsVirial, runningThreads, streamVirial);

    CUDAHELPERS::doReduction1D<dataperthread>(
        thrust::raw_pointer_cast(cudaCoordination.data()),
        thrust::raw_pointer_cast(reductionMemoryCoord.data()), N, nGroups,
        runningThreads, streamCoordination);

    if (nGroups == 1) {
      CUDAHELPERS::plmdDataFromGPU(reductionMemoryVirial, virial, streamVirial);
      // TODO:find a way to stream this
      coordination = reductionMemoryCoord[0];
    } else {
      reductionMemoryVirial.swap(cudaVirial);
      reductionMemoryCoord.swap(cudaCoordination);
    }

    N = nGroups;
  }

  // in this way we do not resize with additional memory allocation
  if (reductionMemoryCoord.size() > cudaCoordination.size())
    reductionMemoryCoord.swap(cudaCoordination);
  if (reductionMemoryVirial.size() > cudaVirial.size())
    reductionMemoryVirial.swap(cudaVirial);
  // this ensures that the memory is fully in the host ram
  hipDeviceSynchronize();
  for (unsigned i = 0; i < deriv.size(); ++i)
    setAtomsDerivatives(i, deriv[i]);

  setValue(coordination);
  setBoxDerivatives(virial);
}
#undef getCoordOrthoPBC
#undef getCoordNoPBC

} // namespace colvar
} // namespace PLMD
